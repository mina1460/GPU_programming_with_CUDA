#include "hip/hip_runtime.h"
#include <iostream>
#include <string>
#include "time_helper.h"
#include <hip/hip_runtime.h>
#define cimg_display 0
#define cimg_use_jpeg
#include "CImg.h"
using namespace cimg_library;

using namespace std;


#define MAX_MASK_SIZE 9
#define MAX_MASK_HEIGHT 3
#define MAX_MASK_WIDTH 3

__constant__ float d_mask[MAX_MASK_SIZE];

#define TILE_SIZE 3
#define BLOCK_SIZE ( TILE_SIZE + MAX_MASK_WIDTH - 1 )

#define cudaCheckError() {                                                                  \
 hipError_t e=hipGetLastError();                                                          \
 if(e!=hipSuccess) {                                                                       \
   printf("Cuda failure %s:%d: '%s'\n",__FILE__,__LINE__,hipGetErrorString(e));            \
   exit(0);                                                                                 \
 }}    

const float blur_kernel[3][3] = {
    {0.0625, 0.125, 0.0625},
    {0.125, 0.25, 0.125},
    {0.0625, 0.125, 0.0625}
};

const float emboss_kernel[3][3] = {
    {-2, -1, 0},
    {-1, 1, 1},
    {0, 1, 2}
};

const float outline_kernel[3][3] = {
    {-1, -1, -1},
    {-1, 8, -1},
    {-1, -1, -1}
};

const float sharpen_kernel[3][3] = {
    {0, -1, 0},
    {-1, 5, -1},
    {0, -1, 0}
};

const float left_sobel_kernel[3][3] = {
    {1, 0, -1},
    {2, 0, -2},
    {1, 0, -1}
};

const float right_sobel_kernel[3][3] = {
    {-1, 0, 1},
    {-2, 0, 2},
    {-1, 0, 1}
};

const float top_sobel_kernel[3][3] = {
    {1, 2, 1},
    {0, 0, 0},
    {-1, -2, -1}
};

const float bottom_sobel_kernel[3][3] = {
    {-1, -2, -1},
    {0, 0, 0},
    {1, 2, 1}
};

// array of pointers to the kernels
const float *kernels[8] = {
    blur_kernel[0],
    emboss_kernel[0],
    outline_kernel[0],
    sharpen_kernel[0],
    left_sobel_kernel[0],
    right_sobel_kernel[0],
    top_sobel_kernel[0],
    bottom_sobel_kernel[0],
};
const string kernel_names[8] = {
    "blur",
    "emboss",
    "outline",
    "sharpen",
    "left_sobel",
    "right_sobel",
    "top_sobel",
    "bottom_sobel",
};

__global__ void convolution_2D_tiled_kernel(float *d_input, float *d_output, int width, int height) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int mask_width = MAX_MASK_WIDTH;
    __shared__ float N_ds[TILE_SIZE][TILE_SIZE];
    if (row < height && col < width) {
        N_ds[threadIdx.y][threadIdx.x] = d_input[row * width + col];
    } 
    __syncthreads();

    int tile_row_start = blockIdx.y * blockDim.y ;
    int tile_row_end = tile_row_start +  blockDim.y;
    int tile_col_start = blockIdx.x * blockDim.x ;
    int tile_col_end = tile_col_start+ blockDim.x;

    if(threadIdx.x == 3 && threadIdx.y == 2 && blockIdx.x == 0 && blockIdx.y == 0) {
        printf("tile_row_start: %d, tile_row_end: %d, tile_col_start: %d, tile_col_end: %d\n", tile_row_start, tile_row_end, tile_col_start, tile_col_end);
        //printing blockdim 
        printf("blockDim.x: %d, blockDim.y: %d\n", blockDim.x, blockDim.y);
    }

    float pvalue = 0;
    int row_start = row - (mask_width / 2);
    int col_start = col - (mask_width / 2);

    for(int i = 0; i < mask_width; i++) {
        for(int j = 0; j < mask_width; j++) {
            int cur_row = row_start + i;
            int cur_col = col_start + j;
            if (cur_row < 0){
                cur_row = 0;
            }  
            else if(cur_row >= height){
                cur_row = height - 1;
            }
            
            if (cur_col < 0){
                cur_col = 0;
            }  
            else if(cur_col >= width){
                cur_col = width - 1;
            }

            int x = threadIdx.x + (cur_col - tile_col_start);
            int y = threadIdx.y + (cur_row - tile_row_start); 

            
            if(cur_row >= tile_row_start && cur_row < tile_row_end 
            && cur_col >= tile_col_start && cur_col < tile_col_end 
            && x < TILE_SIZE && y < TILE_SIZE && x>=0 && y>=0) {
                if(threadIdx.x == 0 && threadIdx.y == 0 && blockIdx.x == 0 && blockIdx.y == 0) {
                printf("cur_row: %d, cur_col: %d, x: %d, y: %d \n", cur_row, cur_col, x, y);
                //printing row and column start
                printf("row_start: %d, col_start: %d\n", row_start, col_start);
                //print i and j 
                printf("i: %d, j: %d\n", i, j);
            }
                pvalue += N_ds[y][x] * d_mask[i * mask_width + j];
            } 
            else if(cur_row < height && cur_col < width) {
                // uses general caching 
                pvalue += d_input[cur_row * width + cur_col] * d_mask[i * mask_width + j];
            }
        }


        if(row * width + col < width*height)
                d_output[row * width + col] = pvalue;
    }
}

__global__ void convolution_output_tiling_2D_kernel(float *d_input, float *d_output, int width, int height) {
    int tx = threadIdx.x;
    int ty = threadIdx.y;
    
    int row_o = blockIdx.y * TILE_SIZE + ty; 
    int col_o = blockIdx.x * TILE_SIZE + tx;
    
    int row_i = row_o - MAX_MASK_WIDTH/2; 
    int col_i = col_o - MAX_MASK_WIDTH/2;
    // this is the same block width = tile size + mask width - 1
    __shared__ float N_ds[TILE_SIZE+MAX_MASK_WIDTH-1][TILE_SIZE+MAX_MASK_HEIGHT-1];
    // grid size is width / tile size
    if(row_i < 0) row_i = 0;
    if(row_i >= height) row_i = height - 1;
    if(col_i < 0) col_i = 0;
    if(col_i >= width) col_i = width - 1;

    N_ds[ty][tx] = d_input[row_i*width+col_i];
    __syncthreads();

    float output = 0.0f;
    if(ty < TILE_SIZE && tx < TILE_SIZE){
        for(int i = 0; i < MAX_MASK_WIDTH; i++) { 
            for(int j = 0; j < MAX_MASK_WIDTH; j++) {
                output += d_mask[i*MAX_MASK_WIDTH+j] * N_ds[i+ty][j+tx]; 
            }
        }
    
        if(row_o < height && col_o < width){ 
            d_output[row_o*width + col_o] = output;
        } 
    }
    return;
}
    
    
void GPU_apply_convolution_kernel(float* h_image, int img_width, int img_height, const float *h_mask, float* output, float *output_2) {

    float *d_image;
    float *d_kernel;
    float *d_result;
    float *d_result_2;
    
    hipMalloc((void **)&d_image, img_width * img_height* sizeof(float));
    cudaCheckError();
    
    // copy filter to symbol memory
    hipMalloc((void **)&d_result, img_width * img_height * sizeof(float));
    cudaCheckError();

    hipMalloc((void **)&d_result_2, img_width * img_height * sizeof(float));
    cudaCheckError();

    // copy image to device
    hipMemcpy(d_image, h_image, img_width * img_height * sizeof(float), hipMemcpyHostToDevice);
    cudaCheckError();

    // and copy filter to static memory
    hipMemcpyToSymbol(HIP_SYMBOL(d_mask), h_mask, 9 * sizeof(float));
    cudaCheckError();

    
    dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
    dim3 dimGrid(ceil((float)img_width / TILE_SIZE), ceil((float)img_height / TILE_SIZE));
    convolution_output_tiling_2D_kernel<<<dimGrid, dimBlock>>>(d_image, d_result, img_width,img_height);
    hipDeviceSynchronize();
    cudaCheckError();

    dim3 dimBlock_generalCache(TILE_SIZE, TILE_SIZE);
    convolution_2D_tiled_kernel<<<dimGrid, dimBlock_generalCache>>>(d_image, d_result_2, img_width,img_height);
    hipDeviceSynchronize();
    cudaCheckError();
    
    // copy result back to host
    
    hipMemcpy(output, d_result, img_width * img_height * sizeof(float), hipMemcpyDeviceToHost);
    cudaCheckError();

    hipMemcpy(output_2, d_result_2, img_width * img_height * sizeof(float), hipMemcpyDeviceToHost);
    cudaCheckError();
    
    hipFree(d_image); 
    cudaCheckError();
    
    hipFree(d_result);
    cudaCheckError();
    
    hipFree(d_result_2);
    cudaCheckError();
    return;
}


void apply_convolution_kernel(float* input_data, float* output_data, const float* kernel, int kernel_width, int kernel_height, int img_width, int img_height)
{
    int output_index, input_r, input_c, input_index, kernel_index;
    float p_value;
    for(int pixel_r=0; pixel_r < img_height; pixel_r++){
        for(int pixel_c=0; pixel_c < img_width; pixel_c++){
            output_index = pixel_r * img_width + pixel_c;
            p_value = 0;
            for(int kernel_r=0; kernel_r < kernel_height; kernel_r++){
                for(int kernel_c=0; kernel_c < kernel_width; kernel_c++){
                
                    input_r = pixel_r + kernel_r - kernel_height/2;
                    input_c = pixel_c + kernel_c - kernel_width/2;
                    
                    if(input_r < 0) input_r = 0;
                    else if(input_r >= img_height) input_r = img_height - 1;
    
                    if(input_c < 0) input_c = 0;   
                    else if(input_c >= img_width) input_c = img_width - 1;
                    
                    input_index = input_r * img_width + input_c;
                    kernel_index = kernel_r * kernel_width + kernel_c;
                    
                    p_value += input_data[input_index] * kernel[kernel_index];
                }
            }
            output_data[output_index] = p_value;
        }
    }
}

bool compare_with_tolerance(float* a, float* b, int size, float tolerance){
    for(int i=0; i < size; i++){
        if(abs(a[i] - b[i]) > tolerance){
            cout << "a[" << i << "] = " << a[i] << " b[" << i << "] = " << b[i] << endl;
            return false;
        }
    }
    return true;
}


int main(int argc, char *argv[])
{
    if(argc != 2)
    {
        cout << "Usage: " << argv[0] << " <img_file_path>" << endl;
        return -1;
    }

    string img_path = argv[1];

    // multiline c++ string 
    const char *  choose_conv_kernel_msg = 
    R"(
        enter a number to choose a convolution kernel:
        1. Blur 
        2. Emboss
        3. Outline
        4. Sharpen
        5. Left Sobel
        6. Right Sobel
        7. Top Sobel
        8. Bottom Sobel
    )";

    cout << choose_conv_kernel_msg << endl;
    int kernel_choice(1);
    cin >> kernel_choice;


    if (kernel_choice < 1 || kernel_choice > 8)
    {
        cout << "invalid kernel choice" << endl;
        return -2;
    }
    kernel_choice -= 1;
    // get the kernel pointer
    const float *kernel = kernels[kernel_choice];

    // open the image with cimg library
    CImg<float> img(img_path.c_str());

    // get the image dimensions
    int width = img.width();
    int height = img.height();
    int depth = img.depth();

    cout << "image dimensions: " << width << "x" << height << "x" << depth << endl;

    string img_basename = img_path.substr(img_path.find_last_of("/\\") + 1);
    cout << "image basename: " << img_basename << endl;
    
    // create a new image to store the result
    float* orig_values = img.data();
    float* result_values = new float[width * height * depth];
    
    cout << "Applying kernel " << kernel_names[kernel_choice] << endl;
    
    for(int i=0; i < width * height * depth; i++){
        result_values[i] = 0;
    }
    
    // apply the convolution kernel
    apply_convolution_kernel(orig_values, result_values, kernel, 3, 3, width, height);
    
    // create a new image to store the result
    CImg<float> result_img(result_values, width, height, 1, depth);
    string result_img_path = img_basename + "_" + kernel_names[kernel_choice] + ".jpeg";
    result_img.save(result_img_path.c_str());
    cout << "result image saved to " << result_img_path << endl;    
    
    float* gpu_result_values = new float[width * height * depth];
    float* gpu_result_values_2 = new float[width * height * depth];
    GPU_apply_convolution_kernel(img.data(),img.width(), img.height(), kernel, gpu_result_values,gpu_result_values_2);

    CImg<float> gpu_result_img(gpu_result_values, width, height, 1, depth);
    
    CImg<float> gpu_result_img_2(gpu_result_values_2, width, height, 1, depth);

    string gpu_result_img_path = img_basename + "_" + kernel_names[kernel_choice] + "_gpu.jpeg";
    string gpu_result_img_path_2 = img_basename + "_" + kernel_names[kernel_choice] + "_gpu_2.jpeg";
    gpu_result_img.save(gpu_result_img_path.c_str());
    cout << "gpu result image saved to " << gpu_result_img_path << endl;
    gpu_result_img_2.save(gpu_result_img_path_2.c_str());
    cout << "gpu result_2 image saved to " << gpu_result_img_path << endl;


    bool res = compare_with_tolerance(result_values, gpu_result_values, width * height * depth, 1e-3);
    if(res){
        cout << "CPU is the same as GPU results\n\n";
    }
    else{
        cout << "Failed!!! CPU is different from GPU\n\n";
    }

    bool res2 = compare_with_tolerance(result_values, gpu_result_values_2, width * height * depth, 1e-3);
    if(res2){
        cout << "CPU is the same as GPU results\n";
    }
    else{
        cout << "Failed!!! CPU is different from GPU\n";
    }
    delete[] result_values;
    return 0;
}
